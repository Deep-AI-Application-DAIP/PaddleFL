#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "conv_op.h"

#include "paddle/fluid/framework/op_registry.h"

#include "core/common/paddle_tensor_impl.cu.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void cu_copy(T* dst, const T* src, size_t numel) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    while (col < numel) {
        dst[col] = src[col];
        col += blockDim.x * gridDim.x;
    }
}


template <typename T>
struct CopyData<platform::CUDADeviceContext, T> {
    void operator()(T* dst, const T* src, size_t numel) {
        dim3 block_size = dim3(PFL_CUDA_THREAD_SIZE, 1);
        dim3 grid_size = dim3((numel + PFL_CUDA_THREAD_SIZE - 1) / PFL_CUDA_THREAD_SIZE, 1);

        cu_copy<T><<<grid_size, block_size, 0, mpc::AbstractContext::_s_stream>>>(
            dst, src, numel);
    }
};

}// namespace paddle
}// namespace operators

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    mpc_conv2d, ops::GemmConvKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    mpc_conv2d_grad,
    ops::GemmConvGradKernel<paddle::platform::CUDADeviceContext, int64_t>);
